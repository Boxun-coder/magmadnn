#include "hip/hip_runtime.h"
/**
 * @file product_internal_device.cu
 * @author Daniel Nichols
 * @version 0.1
 * @date 2019-05-21
 * 
 * @copyright Copyright (c) 2019
 */
#include "compute/product/product_internal.h"

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_product_full_device(T alpha, T *a, T *b, T *out, unsigned int arr_size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < arr_size; i++) {
        out[i] = a[i] * b[i];
    }
}

template <typename T>
void product_full_device(T alpha, Tensor<T> *a, Tensor<T> *b, Tensor<T> *out) {
    kernel_product_full_device <<< 1, a->get_size() >>> (alpha, a->get_ptr(), b->get_ptr(), out->get_ptr(), a->get_size());
}

}   // namespace op
}   // namespace magmadnn