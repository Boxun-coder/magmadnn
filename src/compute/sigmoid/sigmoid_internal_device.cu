#include "hip/hip_runtime.h"
/**
 * @file sigmoid_internal_device.cu
 * @author Daniel Nichols
 * @version 1.0
 * @date 2019-02-23
 *
 * @copyright Copyright (c) 2019
 */
#include "compute/sigmoid/sigmoid_internal.h"
#include "magmadnn/math.h"

#define BLK_SIZE 1024

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_fast_sigmoid_full_device(unsigned int size, T *x, T *out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = x[i] / (1 + abs(x[i]));
    }
}

template <typename T>
__global__ void kernel_sigmoid_full_device(unsigned int size, T *x, T *out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = 1 / (1 + exp(-x[i]));
    }
}

/* exp(INT_TYPE) is not defined in CUDA, so just use 1/(1+|x|) for int.
   Everything will be zero anyways. TODO: decide what to do with int sigmoid. */
template <>
__global__ void kernel_sigmoid_full_device(unsigned int size, int *x, int *out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = 1 / (1 + abs(x[i]));
    }
}

template <typename T>
void sigmoid_full_device(Tensor<T> *x, Tensor<T> *out, bool fast) {

   unsigned int size = out->get_size();
   const auto grid_dim = ceildiv(size, BLK_SIZE);

   if (fast)
      kernel_fast_sigmoid_full_device
         <<<grid_dim, BLK_SIZE>>>
         (size, x->get_ptr(), out->get_ptr());
   else
      kernel_sigmoid_full_device
         <<<grid_dim, BLK_SIZE>>>
         (size, x->get_ptr(), out->get_ptr());
}

template <>
void sigmoid_full_device(Tensor<int> *x, Tensor<int> *out, bool fast) {
    /* sigmoid doesn't make much sense on integer precision */
    for (unsigned int i = 0; i < x->get_size(); i++) x->set(i, (int) exp(x->get(i)));
}

template void sigmoid_full_device(Tensor<float> *x, Tensor<float> *out, bool fast);
template void sigmoid_full_device(Tensor<double> *x, Tensor<double> *out, bool fast);

template <typename T>
void sigmoid_full_device(
      hipStream_t custream, Tensor<T> *x, Tensor<T> *out, bool fast) {

   unsigned int size = out->get_size();
   const auto grid_dim = ceildiv(size, BLK_SIZE);

   if (fast)
      kernel_fast_sigmoid_full_device
         <<<grid_dim, BLK_SIZE, 0, custream>>>
         (size, x->get_ptr(), out->get_ptr());
   else
      kernel_sigmoid_full_device
         <<<grid_dim, BLK_SIZE, 0, custream>>>
         (size, x->get_ptr(), out->get_ptr());
}

template <>
void sigmoid_full_device(hipStream_t custream, Tensor<int> *x, Tensor<int> *out, bool fast) {
    /* sigmoid doesn't make much sense on integer precision */
    for (unsigned int i = 0; i < x->get_size(); i++) x->set(i, (int) exp(x->get(i)));
}

template void sigmoid_full_device(hipStream_t custream, Tensor<float> *x, Tensor<float> *out, bool fast);
template void sigmoid_full_device(hipStream_t custream, Tensor<double> *x, Tensor<double> *out, bool fast);

template <typename T>
__global__ void kernel_sigmoid_grad_device(T *output, T *grad, T *out, unsigned int size, bool is_grad_scalar) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = grad[(is_grad_scalar) ? 0 : i] * output[i] * (1 - output[i]);
    }
}

template <typename T>
void sigmoid_grad_device(Tensor<T> *output, Tensor<T> *grad, Tensor<T> *out) {

   unsigned int size = out->get_size();
   const auto grid_dim = ceildiv(size, BLK_SIZE);

   kernel_sigmoid_grad_device
      <<<grid_dim, BLK_SIZE>>>
      (output->get_ptr(), grad->get_ptr(), out->get_ptr(), size,
       (grad->get_size() == 1));
}
template void sigmoid_grad_device(Tensor<int> *output, Tensor<int> *grad, Tensor<int> *out);
template void sigmoid_grad_device(Tensor<float> *output, Tensor<float> *grad, Tensor<float> *out);
template void sigmoid_grad_device(Tensor<double> *output, Tensor<double> *grad, Tensor<double> *out);

template <typename T>
void sigmoid_grad_device(
      hipStream_t custream, Tensor<T> *output, Tensor<T> *grad, Tensor<T> *out) {

   unsigned int size = out->get_size();
   const auto grid_dim = ceildiv(size, BLK_SIZE);

   kernel_sigmoid_grad_device
      <<<grid_dim, BLK_SIZE, 0, custream>>>
      (output->get_ptr(), grad->get_ptr(), out->get_ptr(), size,
       (grad->get_size() == 1));
}
template void sigmoid_grad_device(hipStream_t custream, Tensor<int> *output, Tensor<int> *grad, Tensor<int> *out);
template void sigmoid_grad_device(hipStream_t custream, Tensor<float> *output, Tensor<float> *grad, Tensor<float> *out);
template void sigmoid_grad_device(hipStream_t custream, Tensor<double> *output, Tensor<double> *grad, Tensor<double> *out);
   
}  // namespace internal
}  // namespace magmadnn

#undef BLK_SIZE
