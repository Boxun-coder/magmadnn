#include "hip/hip_runtime.h"
/**
 * @file sigmoid_internal_device.cu
 * @author Daniel Nichols
 * @version 0.0.1
 * @date 2019-02-23
 * 
 * @copyright Copyright (c) 2019
 */
 #include "compute/sigmoid/sigmoid_internal.h"

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_fast_sigmoid_full_device(unsigned int size, T *x) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	for (unsigned int i = idx; i < size; i += stride) {
        x[i] = x[i] / (1 + abs(x[i]));
	}
}

template <typename T>
__global__ void kernel_sigmoid_full_device(unsigned int size, T *x) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	for (unsigned int i = idx; i < size; i += stride) {
        x[i] = 1 / (1 + exp(-x[i]));
	}
}


/* exp(INT_TYPE) is not defined in CUDA, so just use 1/(1+|x|) for int.
   Everything will be zero anyways. TODO: decide what to do with int sigmoid. */
template <>
__global__ void kernel_sigmoid_full_device(unsigned int size, int *x) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	for (unsigned int i = idx; i < size; i += stride) {
        x[i] = 1 / (1 + abs(x[i]));
	}
}


template <typename T>
void sigmoid_full_device(Tensor<T> *x, bool fast) {
    if (fast)
        kernel_fast_sigmoid_full_device <<<x->get_size(), 1>>> (x->get_size(), x->get_ptr());
    else
        kernel_sigmoid_full_device <<<x->get_size(), 1>>> (x->get_size(), x->get_ptr());
}

template<> void sigmoid_full_device(Tensor<int> *x, bool fast) {
	/* sigmoid doesn't make much sense on integer precision */
	for (unsigned int i = 0; i < x->get_size(); i++)
		x->set(i, (int) exp(x->get(i)));
}

template void sigmoid_full_device(Tensor<float> *x, bool fast);
template void sigmoid_full_device(Tensor<double> *x, bool fast);

}   // namespace internal
}   // namespace magmadnn
