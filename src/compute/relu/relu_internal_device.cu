#include "hip/hip_runtime.h"
/**
 * @file relu_internal_device.cu
 * @author Daniel Nichols
 * @version 0.1
 * @date 2019-05-01
 * 
 * @copyright Copyright (c) 2019
 */
 #include "compute/relu_internal.h"

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_relu_full_device(unsigned int size, T *arr) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < size; i += stride) {
        if (arr[i] < 0) arr[i] = 0;
    }
}

template <typename T>
void relu_full_device(Tensor<T> *x) {
    kernel_relu_full_device <<< x->get_size(), 1 >>> (x->get_size(), x->get_ptr());
}
template void relu_full_device(Tensor<int> *x);
template void relu_full_device(Tensor<float> *x);
template void relu_full_device(Tensor<double> *x);

}   // internal
}   // magmadnn