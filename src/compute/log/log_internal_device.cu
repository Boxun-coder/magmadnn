#include "hip/hip_runtime.h"

#include "compute/log/log_internal.h"

namespace magmadnn {
namespace internal {
 
template <typename T>
__global__ void kernel_log_full_device(T *x, T *out, unsigned int size) {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = log( x[i] );
    }
}
template __global__ void kernel_log_full_device(int *x, int *out, unsigned int size) {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = (int) log( (float) x[i] );
    }
}

template <typename T>
void log_full_device(Tensor<T> *x, Tensor<T> *out) {
    unsigned int size = x->get_size();
    kernel_log_full_device <<< 1, size >>> (x->get_ptr(), out->get_ptr(), size);
}
template void log_full_device(Tensor<int> *x, Tensor<int> *out);
template void log_full_device(Tensor<float> *x, Tensor<float> *out);
template void log_full_device(Tensor<double> *x, Tensor<double> *out);
 
}   // namespace op
}   // namespace magmadnn
