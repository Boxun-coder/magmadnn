#include "hip/hip_runtime.h"
/**
 * @file fill_utilities_device.cu
 * @author Daniel Nichols
 * @version 0.1
 * @date 2019-02-12
 * 
 * @copyright Copyright (c) 2019
 */

#include "fill_utilities_device.h"

#ifdef _HAS_CUDA_

namespace skepsi {

template <typename T>
__global__ void kernel_fill_glorot(T *arr, double *vals) {

}

} // namespace skepsi
#endif